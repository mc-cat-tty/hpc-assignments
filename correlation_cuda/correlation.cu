#include "hip/hip_runtime.h"
#include <iostream>
#include "vector"
#include "cmath"
#include "chrono"
#include "correlation.h"
#include "omp.h"
#include <time.h>
#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif

using namespace std;
using namespace chrono;

template <typename T>
struct Mat
{
    size_t cols_, rows_;
    vector<T> data_;

    Mat(size_t cols, size_t rows) : cols_(cols), rows_(rows)
    {
        data_ = vector<T>(rows * cols, 0);
    }

    T &operator()(size_t r, size_t c)
    {
        return data_[r * cols_ + c];
    }

    const T &operator()(size_t r, size_t c) const
    {
        return data_[r * cols_ + c];
    }

    auto operator&()
    {
        return data_.data();
    }

    size_t size()
    {
        return rows_ * cols_;
    }
    void zeros()
    {
        data_ = vector<T>(rows_ * cols_, 0);
    }
};

template <typename T>
static void init_array(Mat<T> &data)
{
    for (size_t i = 0; i < M; i++)
        for (size_t j = 0; j < N; j++)
            data(i, j) = ((DATA_TYPE)i * j) / M;
}

template <typename T>
static void mean_(Mat<T> &data, vector<T> &mean, T float_n)
{
    for (int j = 0; j < M; j++)
    {
        mean[j] = 0.0;
        for (int i = 0; i < N; i++)
            mean[j] += data(i, j);
        mean[j] /= float_n;
    }
}

template <typename T>
static void stddev_(Mat<T> &data, vector<T> &mean, vector<T> &stddev, T float_n)
{
    DATA_TYPE eps = 0.1f;

    for (size_t j = 0; j < M; j++)
    {
        stddev[j] = 0.0;
        for (size_t i = 0; i < N; i++)

            stddev[j] += (data(i, j) - mean[j]) * (data(i, j) - mean[j]);
        stddev[j] /= float_n;
        stddev[j] = sqrt(stddev[j]);
        /* The following in an inelegant but usual way to handle
           near-zero std. dev. values, which below would cause a zero-
           divide. */
        stddev[j] = stddev[j] <= eps ? 1.0 : stddev[j];
    }
}

template <typename T>
static void center_reduce_(Mat<T> &data, vector<T> &mean, vector<T> &stddev, T float_n)
{
    for (size_t i = 0; i < N; i++)
        for (size_t j = 0; j < M; j++)
        {
            data(i, j) -= mean[j];
            data(i, j) /= sqrt(float_n) * stddev[j];
        }
}

template <typename T>
static void compute_corr_(Mat<T> &data, Mat<T> &symmat, T float_n)
{
    size_t j1, j2, i;
    /* Calculate the m * m correlation matrix. */
    for (j1 = 0; j1 < M - 1; j1++)
    {
        symmat(j1, j1) = 1.0;
        for (j2 = j1 + 1; j2 < M; j2++)
        {
            symmat(j1, j2) = 0.0;
            for (i = 0; i < N; i++)
                symmat(j1, j2) += (data(i, j1) * data(i, j2));
            symmat(j2, j1) = symmat(j1, j2);
        }
    }
    symmat(N - 1, N - 1) = 1.0;
}

template <typename T>
static void compute_corr_loop_interchange_not_optimized_(Mat<T> &data, Mat<T> &symmat, T float_n)
{

    for (size_t j1 = 0; j1 < M - 1; j1++)
    {
        symmat(j1, j1) = 1.0;
        for (size_t j2 = j1 + 1; j2 < M; j2++)
            symmat(j1, j2) = 0.0;
    }

    for (size_t i = 0; i < N; i++)
        for (size_t j1 = 0; j1 < M - 1; j1++)
            for (size_t j2 = j1 + 1; j2 < M; j2++)
                symmat(j1, j2) += (data(i, j1) * data(i, j2));

    for (size_t j1 = 0; j1 < M - 1; j1++)
        for (size_t j2 = j1 + 1; j2 < M; j2++)
            symmat(j2, j1) = symmat(j1, j2);
    symmat(M - 1, M - 1) = 1.0;
}

template <typename T>
static void compute_corr_loop_interchange_task_opt_(Mat<T> &data, Mat<T> &symmat, T float_n)
{
    cout << "warning: for some reason task based seems not working\n";
    size_t i, j1, j2;

#pragma omp task
    for (i = 0; i < N; i++)
        symmat(i, i) = 1.0;

    for (j1 = 0; j1 < M - 1; j1++)
#pragma omp task
#pragma omp simd
        for (j2 = j1 + 1; j2 < M; j2++)
            symmat(j1, j2) = 0.0;

    int unroll_size_ = 4;
    int blocks = N / unroll_size_;
#pragma omp taskwait

    for (size_t i = 0; i < blocks; i += 1)
#pragma omp task
        for (j1 = 0; j1 < M - 1; j1++)
#pragma omp simd
            for (j2 = j1 + 1; j2 < M; j2++)
            {
                size_t idx = i * unroll_size_;
                symmat(j1, j2) += (data(idx, j1) * data(idx, j2));
                symmat(j1, j2) += (data(idx + 1, j1) * data(idx + 1, j2));
                symmat(j1, j2) += (data(idx + 2, j1) * data(idx + 2, j2));
                symmat(j1, j2) += (data(idx + 3, j1) * data(idx + 3, j2));
            }
#pragma omp taskwait

    for (size_t i = unroll_size_ * blocks; i < N; i++)
#pragma omp task
        for (size_t j1 = 0; j1 < M - 1; j1++)
#pragma omp simd
            for (size_t j2 = j1 + 1; j2 < M; j2++)
                symmat(j1, j2) += (data(i, j1) * data(i, j2));
#pragma omp taskwait

    for (size_t j1 = 0; j1 < M - 1; j1++)
#pragma omp task
#pragma omp simd
        for (size_t j2 = j1 + 1; j2 < M; j2++)
            symmat(j2, j1) = symmat(j1, j2);

    symmat(M - 1, M - 1) = 1.0;
}

template <typename T>
static void compute_corr_loop_interchange_parallel_opt_(Mat<T> &data, Mat<T> &symmat, T float_n)
{
#pragma omp parallel for
    for (size_t j1 = 0; j1 < M - 1; j1++)
    {
        symmat(j1, j1) = 1.0;
        for (size_t j2 = j1 + 1; j2 < M; j2++)
            symmat(j1, j2) = 0.0;
    }

    int unroll_size_ = 4;
    int blocks = N / unroll_size_;
    for (size_t i = 0; i < blocks; i += 1)
#pragma omp parallel for schedule(dynamic)
        for (size_t j1 = 0; j1 < M - 1; j1++)
#pragma omp simd
            for (size_t j2 = j1 + 1; j2 < M; j2++)
            {
                size_t idx = i * unroll_size_;
                symmat(j1, j2) += (data(idx, j1) * data(idx, j2));
                symmat(j1, j2) += (data(idx + 1, j1) * data(idx + 1, j2));
                symmat(j1, j2) += (data(idx + 2, j1) * data(idx + 2, j2));
                symmat(j1, j2) += (data(idx + 3, j1) * data(idx + 3, j2));
            }

    for (size_t i = unroll_size_ * blocks; i < N; i++)
        for (size_t j1 = 0; j1 < M - 1; j1++)
            for (size_t j2 = j1 + 1; j2 < M; j2++)
                symmat(j1, j2) += (data(i, j1) * data(i, j2));

#pragma omp parallel for
    for (size_t j1 = 0; j1 < M - 1; j1++)
#pragma omp simd
        for (size_t j2 = j1 + 1; j2 < M; j2++)
            symmat(j2, j1) = symmat(j1, j2);

    symmat(M - 1, M - 1) = 1.0;
}

template <typename T>
ostream &operator<<(ostream &os, const Mat<T> &data)
{
    for (int r = 0; r < data.rows_; ++r)
    {
        for (int c = 0; c < data.cols_; ++c)
        {
            cout << data(r, c) << " ";
        }
        cout << endl;
    }
    return os;
}

template <typename T>
static void hash_(Mat<T> &symmat)
{
    double hash_ = 0.;
    for (size_t i = 0; i < M; i++)
    {
        for (size_t j = 0; j < M; j++)
            hash_ += symmat(i, j);
    }
    printf("The computed hash: %f\n", hash_);
}

struct Timer
{
    time_point<steady_clock> start_ = steady_clock::now();
    time_point<steady_clock> stop_ = steady_clock::now();
    string task_name_;

public:
    explicit Timer(string task_name) : task_name_(task_name) {}

    void start()
    {
        start_ = steady_clock::now();
    }
    void start(string task_name)
    {
        task_name_ = task_name;
        start_ = steady_clock::now();
    }
    void stop()
    {
        stop_ = steady_clock::now();
        duration<double> elapsed_ms = stop_ - start_;
        cout << "Elapsed time for " << task_name_ << ": " << (elapsed_ms.count()) << "s" << endl;
    }
};

template <typename T>
static void kernel_correlation(size_t m, size_t n, DATA_TYPE float_n, Mat<T> &data, Mat<T> &symmat,
                               vector<T> &mean, vector<T> &stddev)
{
    mean_(data, mean, float_n);
    stddev_(data, mean, stddev, float_n);
    center_reduce_(data, mean, stddev, float_n);
    compute_corr_(data, symmat, float_n);
}

template <typename T>
static void kernel_correlation_optimized(size_t m, size_t n, DATA_TYPE float_n, Mat<T> &data, Mat<T> &symmat,
                                         vector<T> &mean, vector<T> &stddev)
{
    Timer t("Corr");

    // t.start("Mean");/
    mean_(data, mean, float_n);
    // t.stop();

    // t.start("Std Deviation");
    stddev_(data, mean, stddev, float_n);
    // t.stop();

    // t.start("Center Reduce");
    center_reduce_(data, mean, stddev, float_n);
    // t.stop();

#ifdef LOOP_OPT
    t.start("Loop Opt Corr");
    compute_corr_loop_interchange_not_optimized_(data, symmat, float_n);
#endif
#ifdef TASK_OPT
    t.start("Task Opt Corr");
#pragma omp parallel
    {
#pragma omp master
        compute_corr_loop_interchange_task_opt_(data, symmat, float_n);
    }
#endif
#ifdef PARALLEL_OPT
    t.start("parallel Opt Corr");
    compute_corr_loop_interchange_parallel_opt_(data, symmat, float_n);
#endif
    t.stop();
    // compute_corr_(data, symmat, float_n);
}

template <typename T>
__global__ void corr_kernel_(T *data, T *symmat, size_t height, size_t width)
{
    const size_t row = threadIdx.y + blockDim.y * blockIdx.y;
    const size_t col = threadIdx.x + blockDim.x * blockIdx.x;
    if (row < (height) and col < (width))
    {
        for (size_t j2 = col + 1; j2 < width; j2++)
            // symmat[col * width + j2] += (data[row * width + col] * data[row * width + j2]);
            atomicAdd(&(symmat[col * width + j2]), (data[row * width + col] * data[row * width + j2]));
    }
}

template <typename T>
__global__ void corr_kernel_2_(T *data, T *symmat, size_t height, size_t width)
{

    const size_t row = threadIdx.y + blockDim.y * blockIdx.y;
    const size_t col = threadIdx.x + blockDim.x * blockIdx.x;
    const T v = data[row * width + col];
    size_t col_start = threadIdx.x + 1;
    __shared__ T data_shr[BLOCK_SIZE][BLOCK_SIZE];
    if (row < (height) and col < (width))
    {

        for (size_t i = blockIdx.x; i < (width - 1 + BLOCK_SIZE) / BLOCK_SIZE; i++)
        {
            size_t col_offset = threadIdx.x + i * BLOCK_SIZE;

            data_shr[threadIdx.y][threadIdx.x] = (col_offset < width) ? data[row * width + col_offset] : 0;
            __syncthreads();

            size_t upper_bound = (i == (((width - 1 + BLOCK_SIZE) / BLOCK_SIZE) - 1)) ? width - i + BLOCK_SIZE : BLOCK_SIZE;
            for (size_t j = col_start; j < BLOCK_SIZE and ( i * BLOCK_SIZE + j) < width; j++)
            {
                atomicAdd(&(symmat[col * width + (j + i * blockDim.x)]), (v * data_shr[threadIdx.y][j]));
            }
            col_start = 0;
            __syncthreads();
        }
    }
}

int main()
{
    DATA_TYPE float_n = 1.2;
    Mat<DATA_TYPE> data(M, N);
    init_array(data);
    vector<DATA_TYPE> mean(M, 0);
    vector<DATA_TYPE> stddev(M, 0);
    Mat<DATA_TYPE> symmat(M, M);
    Timer t("correlation");
    struct timespec rt[2];

#ifdef BASELINE
    t.start("baseline correlation");
    kernel_correlation(M, N, float_n, data, symmat, mean, stddev);
    t.stop();
    hash_(symmat);
    mean = vector<DATA_TYPE>(M, 0);
    stddev = vector<DATA_TYPE>(M, 0);
    symmat.zeros();
    init_array(data);
#elif defined(LOOP_OPT) or defined(TASK_OPT) or defined(PARALLEL_OPT)
    t.start("Correlation Optimized");
    kernel_correlation_optimized(M, N, float_n, data, symmat, mean, stddev);
    t.stop();
    hash_(symmat);
#endif

#ifdef CUDA
    symmat.zeros();
    clock_gettime(CLOCK_REALTIME, rt + 0);
    DATA_TYPE *data_d, *symmat_d;
    hipMalloc((void **)&data_d, sizeof(DATA_TYPE) * M * N);
    hipMalloc((void **)&symmat_d, sizeof(DATA_TYPE) * N * N);

    hipMemcpy(data_d, &data, sizeof(DATA_TYPE) * M * N, hipMemcpyHostToDevice);
    hipMemset(symmat_d, 0, sizeof(DATA_TYPE) * N * N);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim(((N - 1 + BLOCK_SIZE) / BLOCK_SIZE), ((M - 1 + BLOCK_SIZE) / BLOCK_SIZE));
    t.start("GPU CORR");
    corr_kernel_2_<<<gridDim, blockDim>>>(data_d, symmat_d, M, N);
    hipMemcpy(&symmat, symmat_d, sizeof(DATA_TYPE) * M * N, hipMemcpyDeviceToHost);
    t.stop();

    clock_gettime(CLOCK_REALTIME, rt + 1);
    double wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
    for (size_t j1 = 0; j1 < M - 1; j1++)
    {
        symmat(j1, j1) = 1.0;
        for (size_t j2 = j1 + 1; j2 < M; j2++)
            symmat(j2, j1) = symmat(j1, j2);
    }
    symmat(M - 1, M - 1) = 1.0;
    hash_(symmat);
    printf("corr_kernel_ (GPU): %9.3f sec %9.1f GFLOPS\n", wt, (float)N * (M - 1) * M / (1.0e9 * wt * 2.0));
#endif

    return 0;
}
